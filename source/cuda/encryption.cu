#include "hip/hip_runtime.h"
#include "encryption.cuh"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/hip_runtime.h>

__global__ void hash(const std::uint64_t* const values, std::uint64_t* const hashes, const unsigned int length) {
    __shared__ constexpr auto val_a = std::uint64_t{ 5'647'095'006'226'412'969 };
    __shared__ constexpr auto val_b = std::uint64_t{ 41'413'938'183'913'153 };
    __shared__ constexpr auto val_c = std::uint64_t{ 6'225'658'194'131'981'369 };
    int x_global = blockIdx.x * blockDim.x + threadIdx.x;
    if(x_global < length){
        std::uint64_t value = values[x_global];
        const auto val_1 = (value >> 14) + val_a;
        const auto val_2 = (value << 54) ^ val_b;
        const auto val_3 = (val_1 + val_2) << 4;
        const auto val_4 = (value % val_c) * 137;

        hashes[x_global] = val_3 ^ val_4;
        //return final_hash;
    }
}

__global__ void flat_hash(const std::uint64_t* const values, std::uint64_t* const hashes, const unsigned int length){
    __shared__ constexpr auto val_a = std::uint64_t{ 5'647'095'006'226'412'969 };
    __shared__ constexpr auto val_b = std::uint64_t{ 41'413'938'183'913'153 };
    __shared__ constexpr auto val_c = std::uint64_t{ 6'225'658'194'131'981'369 };
    int x_global = blockIdx.x * blockDim.x + threadIdx.x;

    //New Added for loop to account for case length > blockDim.x
    for(int i = x_global; i < length; i+=blockDim.x){
        if(x_global < length){
            std::uint64_t value = values[x_global];
            const auto val_1 = (value >> 14) + val_a;
            const auto val_2 = (value << 54) ^ val_b;
            const auto val_3 = (val_1 + val_2) << 4;
            const auto val_4 = (value % val_c) * 137;

            hashes[i] = val_3 ^ val_4;
        }
    }
}
__global__ void find_hash(const std::uint64_t* const hashes, unsigned int* const indices, const unsigned int length, const std::uint64_t searched_hash, unsigned int* const ptr){
    int x_global = blockIdx.x * blockDim.x + threadIdx.x;

    if (x_global < length && searched_hash == hashes[x_global]) {

        // We are sorry, that you have to read this ;D

        // First the value in pointer is incremented by one
        // The current index is the new pointer value - 1,
        // since the pointer value is set to 1 by the first accessing thread (and it should start at 0)

        // Finally the indices value is updated at the correct position!! Yeaa!
        indices[atomicAdd(*ptr, 1)-1] = x_global;
    }
}

__global__ void hash_schemes (std::uint64_t* const hashes, const unsigned int length) {
    __shared__ constexpr auto val_a = std::uint64_t{ 5'647'095'006'226'412'969 };
    __shared__ constexpr auto val_b = std::uint64_t{ 41'413'938'183'913'153 };
    __shared__ constexpr auto val_c = std::uint64_t{ 6'225'658'194'131'981'369 };
    int x_global = blockIdx.x * blockDim.x + threadIdx.x;
    if (x_global < length) {
        std::uint64_t value = x_global;
        value += value << 32;
        const auto val_1 = (value >> 14) + val_a;
        const auto val_2 = (value << 54) ^ val_b;
        const auto val_3 = (val_1 + val_2) << 4;
        const auto val_4 = (value % val_c) * 137;
        hashes[x_global] = val_3 ^ val_4;
    }
}

