#include "hip/hip_runtime.h"
#include "image.cuh"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

// Task 2b)
__global__ void grayscale_kernel(const Pixel<std::uint8_t>* const input, Pixel<std::uint8_t>* const output, const unsigned int width, const unsigned int height) {
  int x_global = blockIdx.x * blockDim.x + threadIdx.x;
  int y_global = blockIdx.y * blockDim.y + threadIdx.y;

  if (y_global < height && x_global < width) {
    const auto pixel = input[y_global * height + x_global];
    const auto r = pixel.get_red_channel();
    const auto g = pixel.get_green_channel();
    const auto b = pixel.get_blue_channel();
  
    const auto gray = r * 0.2989 + g * 0.5870 + b * 0.1140;
    const auto gray_converted = static_cast<std::uint8_t>(gray);
  
    const auto gray_pixel = BitmapPixel{ gray_converted , gray_converted,  gray_converted };
  
    output[y_global * height + x_global] = gray_pixel;
  }
}

BitmapImage get_grayscale_cuda(const BitmapImage& source);
