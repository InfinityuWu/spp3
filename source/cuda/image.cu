#include "hip/hip_runtime.h"
#include "image.cuh"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "common.cuh"

// Task 2b)
__global__ void grayscale_kernel (const Pixel<std::uint8_t>* const input, Pixel<std::uint8_t>* const output, const unsigned int width, const unsigned int height) {
  int x_global = blockIdx.x * blockDim.x + threadIdx.x;
  int y_global = blockIdx.y * blockDim.y + threadIdx.y;

  if (y_global < height && x_global < width) {
    const auto pixel = input[y_global * width + x_global];
    const auto r = pixel.get_red_channel();
    const auto g = pixel.get_green_channel();
    const auto b = pixel.get_blue_channel();
  
    const auto gray = r * 0.2989 + g * 0.5870 + b * 0.1140;
    const auto gray_converted = static_cast<std::uint8_t>(gray);
  
    const auto gray_pixel = BitmapPixel{ gray_converted , gray_converted,  gray_converted };
  
    output[y_global * width + x_global] = gray_pixel;
  }
}

// Task 2c)
BitmapImage get_grayscale_cuda (const BitmapImage& source) {
  auto output_image = BitmapImage{source.get_height(), source.get_width()};
  int number_threads_per_block = 16;
  // creating pointers to be used to work on device (-> GPU) memory
  Pixel<std::uint8_t>* input_gpu;
  Pixel<std::uint8_t>* output_gpu;
  // allocating device memory of required size dictated by dimensions of the image and having the pointers written to the previously established variables
  hipMalloc((void**) &input_gpu, source.get_height() * source.get_width() * sizeof(Pixel<std::uint8_t>));
  hipMalloc((void**) &output_gpu, source.get_height() * source.get_width() * sizeof(Pixel<std::uint8_t>));
  // copying the entire image pixel data from host (-> CPU) to device as input to the kernel to operate on
  hipMemcpy(input_gpu, source.get_data(), source.get_height() * source.get_width() * sizeof(Pixel<std::uint8_t>), hipMemcpyHostToDevice);
  grayscale_kernel<<< {divup(source.get_height(), number_threads_per_block), divup(source.get_width(), number_threads_per_block)}, {number_threads_per_block, number_threads_per_block} >>>(input_gpu, output_gpu, source.get_width(), source.get_height());
  // copying the entire image pixel data back over from the device to host after the kernel has finished its calculation and has arrived at the desired transformation of the input data
  hipMemcpy(output_image.get_data(), output_gpu, source.get_height() * source.get_width() * sizeof(Pixel<std::uint8_t>), hipMemcpyDeviceToHost);
  // freeing device memory
  hipFree(input_gpu);
  hipFree(output_gpu);
  return output_image;
}