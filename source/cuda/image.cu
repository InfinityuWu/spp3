#include "hip/hip_runtime.h"
#include "image.cuh"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "common.cuh"

// Task 2b)
__global__ void grayscale_kernel (const Pixel<std::uint8_t>* const input, Pixel<std::uint8_t>* const output, const unsigned int width, const unsigned int height) {
  int x_global = blockIdx.x * blockDim.x + threadIdx.x;
  int y_global = blockIdx.y * blockDim.y + threadIdx.y;

  if (y_global < height && x_global < width) {
    const auto pixel = input[y_global * width + x_global];
    const auto r = pixel.get_red_channel();
    const auto g = pixel.get_green_channel();
    const auto b = pixel.get_blue_channel();
  
    const auto gray = r * 0.2989 + g * 0.5870 + b * 0.1140;
    const auto gray_converted = static_cast<std::uint8_t>(gray);
  
    const auto gray_pixel = BitmapPixel{ gray_converted , gray_converted,  gray_converted };
  
    output[y_global * width + x_global] = gray_pixel;
  }
}

// Task 2c)
BitmapImage get_grayscale_cuda (const BitmapImage& source) {
  auto output_image = BitmapImage{source.get_height(), source.get_width()};
  int number_threads_per_block = 16;
  Pixel<std::uint8_t>** input_gpu;
  Pixel<std::uint8_t>** output_gpu;
  hipMalloc((void**) &input_gpu, source.get_height() * source.get_width() * sizeof(Pixel<std::uint8_t>));
  hipMalloc((void**) &output_gpu, source.get_height() * source.get_width() * sizeof(Pixel<std::uint8_t>));
  hipMemcpy(input_gpu, source.get_data(), source.get_height() * source.get_width() * sizeof(Pixel<std::uint8_t>), hipMemcpyHostToDevice);
  grayscale_kernel<<< {divup(source.get_height(), number_threads_per_block), divup(source.get_width(), number_threads_per_block)}, {number_threads_per_block, number_threads_per_block} >>>(input_gpu, output_gpu, source.get_width(), source.get_height());
  hipMemcpy(output_image.get_data(), output_gpu, source.get_height() * source.get_width() * sizeof(Pixel<std::uint8_t>), hipMemcpyDeviceToHost);
  return output_image;
}
